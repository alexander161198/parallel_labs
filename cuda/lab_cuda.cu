#include<stdio.h>
#include<iostream> 
#include<stdlib.h> 
#include<math.h> 
#include<cmath> 

#include "hip/hip_runtime.h"


using namespace std; 

#define X 32
#define Y 32 
#define X_IN 10 
#define Y_IN 10

#define N ((X+1) * (Y+1)) 
#define TIME 10

double h_x = 0.2; 
double h_y = 0.2; 
double h_t = 1;


__global__ void Gauss_forward(double *matr, double *slv, int n)
{
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = j + 1; i < n; i++)
	{
		double koef = matr[i*N + j] / matr[j*N + j]; 
		for(int k = j; k < n; k++)
		{
			matr[i*N + k] -= koef * matr[j*N + k]; 
		}

		slv[i] -= koef * slv[j]; 
	}
}


__global__ void Gauss_reverse(double *matr, double *slv, double *solve, int n)
{
	int k = n - 1 - blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0.0;
	for(int j = k + 1; j < n; j++)
	{ 
		sum += solve[j] * matr[k*N + j];
	}

	solve[k] = (slv[k] - sum) / matr[k*N + k]; 
}

int inside_tube(int i, int j, int flag)
{ 
	if(flag)
	{
		if((j >= (X - X_IN) / 2) && (j <= (X + X_IN) / 2) && (i >= (Y - Y_IN) / 2) && (i <= (Y + Y_IN) / 2) )
		{
			return 1; 
		}
		else
		{
			return 0; 
		}
	} 

	else
	{
		if((j > (X - X_IN) / 2) && (j < (X + X_IN) / 2) && (i > (Y - Y_IN) / 2) && (i < (Y + Y_IN) / 2) )
		{
			return 1; 
		}
		else
		{
			return 0; 
		}
	} 
}

double sss[2];

int main()
{
	//FILE* new_file = fopen("stat2", "w");
	double *matr; 
	matr = (double*)malloc(sizeof(double) * N * N);
/*
	timespec start, end;
	clock_gettime(CLOCK_MONOTONIC_RAW, &start);*/

	// заполняем матрицу нулями 
	for(int i = 0; i < N; ++i)
	{ 
		for (int j = 0; j < N; ++j)
		{
			matr[i*N + j] = 0.0; 
		}
	}

	double* solve = new double [N]; 
	double* x = new double [N]; 

	for(int i = 0; i <= Y; ++i)
	{ 
		for (int j = 0; j <= X; ++j)
		{
			if( inside_tube(i, j, 1) )
			{ 
				if(inside_tube(i, j, 0))
				{
				// температура будет решением для уравнения Гаусса во всех точках,
				// кроме пограничных, где заданы ГУ 3 рода 
					solve[i * (X + 1) + j] = 0;
				} 
				else
				{
					solve[i * (X + 1) + j] = 100.0; 
				}
			} 
			else
			{
				solve[i * (X + 1) + j] = 10.0; 
			}
		} 
	}

	// ГУ первого рода на внутренней стенке. Температура в узлах будет сохраняться
	for(int i = (Y - Y_IN) / 2; i <= (Y + Y_IN) / 2; ++i)
	{ 
		for (int j = (X - X_IN) / 2; j <= (X + X_IN) / 2; ++j)
		{
			int k = i * (X + 1) + j; 
			matr[k*N + k] = 1;
		} 
	}

	// ГУ 3го рода на внешних стенках 
	for (int j = 1; j < X; ++j)
	{
		//int i = 0;
		int k_vnesh = j;
		int k_vnytr = j + (X + 1); 
		solve[k_vnesh] = 0; 

		matr[k_vnesh*N + k_vnesh] = 1 / h_y + 1; 
		matr[k_vnesh*N + k_vnytr] = -1 / h_y;
	} 

	for (int j = 1; j < X; ++j)
	{ 
		int i = Y;
		int k_vnesh = j + (X + 1) * i;
		int k_vnytr = j + (X + 1) * (i - 1); 

		solve[k_vnesh] = 0; 
		matr[k_vnesh*N + k_vnesh] = 1 / h_y + 1; 
		matr[k_vnesh*N + k_vnytr] = -1 / h_y;
	}

	for (int i = 0; i <= Y; ++i)
	{ 
		//int j = 0;
		int k_vnesh = i * (X + 1); 
		int k_vnytr = i * (X + 1) + 1; 
		solve[k_vnesh] = 0;

		matr[k_vnesh*N + k_vnesh] = 1 / h_x + 1; 
		matr[k_vnesh*N + k_vnytr] = - 1 / h_x;
	}

	for (int i = 0; i <= Y; ++i)
	{ 
		int j = X;
		int k_vnesh = i * (X + 1) + j;
		int k_vnytr = i * (X + 1) + j - 1; 
		solve[k_vnesh] = 0; 

		matr[k_vnesh*N + k_vnesh] = 1 / h_x + 1; 
		matr[k_vnesh*N + k_vnytr] = - 1 / h_x;
	}

	// Внутренние узлы pаданные с помощью стандартной неявной схемы (по формуле 2)
	for (int i = 1; i < Y; ++i)
	{ 
		for (int j = 1; j < X; ++j)
		{
			if( inside_tube(i, j, 1) )
			{ 
				continue;
			}
			int ij = i * (X + 1) + j;
			int im1j = (i - 1) * (X + 1) + j; 
			int ip1j = (i + 1) * (X + 1) + j; 
			int ijm1 = i * (X + 1) + j - 1; 
			int ijp1 = i * (X + 1) + j + 1;

			matr[ij*N + ij] = 2 * ( h_t / (h_x * h_x) + h_t / (h_y * h_y) ) + 1;
			matr[ij*N + im1j] = - h_t / (h_y * h_y); 
			matr[ij*N + ip1j] = - h_t / (h_y * h_y); matr[ij*N + ijm1] = - h_t / (h_x * h_x); matr[ij*N + ijp1] = - h_t / (h_x * h_x);
			
			solve[ij] = x[ij]; 
		}
	}

	timespec start, end;
	clock_gettime(CLOCK_MONOTONIC_RAW, &start);

	int n = N;
	double *device_matr;
	double *device_slv;
	double *device_solve;
	unsigned int size_matr = sizeof(double) * n * n;
	unsigned int size_slv = sizeof(double) * n;

	hipError_t cudaStatus;

	hipMalloc((void**)&device_matr, size_matr);
	hipMalloc((void**)&device_slv, size_slv);
	hipMalloc((void**)&device_solve, size_slv);

	float timerValueGPU;
	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);	
	
	dim3 N_Treads(8);
	dim3 N_Block(n / 8);

	for (int k = 0; k <= TIME / h_t; k+= h_t) 
	{ 
		cout << k << endl;
		// в solve после решения системы уравнений методом Гаусса буде храниться температура трубы.
		// Чтобы избежать копирования темературы в другой массив, заметим, что в большинстве узлов
		// получення температура является решением для следующей итерации расчётов.
		// Температура не является решением только на границах трубы, где заданы ГУ 3-го рода.
		// В этих узлах в столбце-ответе должен стоять 0, установим его.
		for (int j = 1; j < X; ++j)
		{ 
			solve[j] = 0;
			solve[j + (X + 1) * Y] = 0; 
		}

		for (int i = 0; i <= Y; ++i)
		{ 
			solve[i * (X + 1)] = 0; 
			solve[i * (X + 1) + X] = 0;
		}

		hipMemcpy(device_solve, solve, size_slv, hipMemcpyHostToDevice);	

		double *matr2;
		matr2 = (double*)malloc(sizeof(double) * N * N);

		double *slv = new double[n];

		for (int i = 0; i < N; i++)
		{ 
			for (int j = 0; j < N; j++)
			{
				matr2[i*N + j] = matr[i*N + j]; 
			}
			slv[i] = solve[i]; 
		}

		hipMemcpy(device_matr, matr2, size_matr, hipMemcpyHostToDevice);	
		hipMemcpy(device_slv, slv, size_slv, hipMemcpyHostToDevice);

		Gauss_forward <<< N_Block, N_Treads >>> (device_matr, device_slv, n);	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) 
		{
			cout << "Solve last error:" << hipGetErrorString(cudaStatus) << endl;
			return 0;
		}
		hipDeviceSynchronize();

		hipMemcpy(slv, device_slv, size_slv, hipMemcpyDeviceToHost);
		hipMemcpy(matr2, device_matr, size_matr, hipMemcpyDeviceToHost);

		Gauss_reverse <<< N_Block, N_Treads >>> (device_matr, device_slv, device_solve, n);	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) 
		{
			cout << "Solve last error:" << hipGetErrorString(cudaStatus) << endl;
			return 0;
		}
		hipDeviceSynchronize();

		hipMemcpy(solve, device_solve, size_slv, hipMemcpyDeviceToHost);
		hipMemcpy(slv, device_slv, size_slv, hipMemcpyDeviceToHost);
		hipMemcpy(matr2, device_matr, size_matr, hipMemcpyDeviceToHost);

		delete matr2; 
		delete slv;

		// теперь в solve хранится температура. Записываем её в файл.
/*		for (int i = 0; i <= Y; i++) 
		{
			for (int j = 0; j <= X; j++) 
			{ 
				fprintf(new_file, "%3.2lf ", solve[i * (X + 1) + j]); 
			}

			fprintf(new_file, "\n"); 
		}

		fprintf(new_file, "\n"); */
	}

	hipEventRecord(stop2, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&timerValueGPU, start2, stop2);
	cout << "GPU calculation time " << timerValueGPU << " msec" <<endl;

	hipFree(device_matr);
	hipFree(device_slv);
	hipFree(device_solve);

	clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    printf("Time taken: %lf sec.\n",end.tv_sec-start.tv_sec+ 0.000000001*(end.tv_nsec-start.tv_nsec));

	//fclose(new_file);

	delete matr; 
	delete solve; 
	delete x; 

	return 0;
} 