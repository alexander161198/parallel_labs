#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <ctime> 
#include <iostream>
#include "inverse.h"
#include "boost/numeric/ublas/matrix.hpp"
#include "boost/numeric/ublas/vector.hpp"
using namespace boost::numeric::ublas;


//граничное условие 2 рода dT/dn=10
#define TT 10

matrix <double> all_calc (double a1,double a2,double L1,double L2,int N_i,int N_j, int k) {
	double x1 = (L1-a1)/2;
    double y1 = (L2-a2)/2; 
    double x2 = (L1+a1)/2; 
    double y2 = (L2+a2)/2; 

    double dx = L1/(N_i - 1);
    double dy = L2/(N_j - 1);

    //Инициализируем начальные массивы

    matrix<double> M (N_i*N_j,N_i*N_j);
    vector<double> R (N_i*N_j);

    for (int i = 0; i < N_i; i++) {
        for (int j = 0; j < N_j; j++) {
            double x=i*dx;
            double y=j*dy;
            if (i==0) { //самая левая граница dT/dn=10
                M(j+N_j*(i),j+N_j*(i+1)) = 1/dx; //i+1, j
                M(j+N_j*(i),j+N_j*(i)) = -1/dx;//i,j
                R(j + N_j * (i)) = TT;
                continue;
            }

            if (j==0){	//нижняя dT/dn=10
            	M(j+N_j*(i),j+N_j*(i)) = -1/dy;//i, j,
                M(j+N_j*(i),j+N_j*(i)+1) = 1/dy;//i,j+1
                R(j + N_j * (i)) = TT;
            	continue;
            }
            if (j==N_j-1) { //%самая верхняя граница dT/dn =10
            	M(j+N_j*(i),j+N_j*(i)) = -1/dy;//i, j,
                M(j+N_j*(i),j+N_j*(i)-1) = 1/dy;//i,j-1
                R(j + N_j * (i)) = TT;
                continue;
            }
            if (i==N_i-1) { //самая правая граница dT/dn=10
            	M(j+N_j*(i), j+N_j*(i-1)) = 1/dx; //i-1, j
                M(j+N_j*(i),j+N_j*(i)) = -1/dx;//i,j
                R(j + N_j * (i)) = TT;
                continue;
            }
            //внутренняя граница нижняя
            if ((y<=y1) &&((y+dy)>y1) && (x<=x2) &&(x>=x1)) { 
                M(j + N_j * (i),j + N_j * (i)) = 1;//i, j
                R(j + N_j * (i)) = 100;
                continue;
            }
            //внутренняя граница верхняя
            if ((y>y2) &&((y-dy)<y2) && (x<=x2) && (x>=x1)) { 
                M(j + N_j * (i),j + N_j * (i)) = 1;//i, j
                R(j + N_j * (i)) = 100;
                continue;
            }
            //внутренняя граница правая
            if (((x-dx)<x2) &&(x>=x2) && (y<=y2) &&(y>=y1)) {
            	M(j + N_j * (i),j + N_j * (i)) = 1;//i, j
                R(j + N_j * (i)) = 100;
                continue;
            }
            //внутренняя граница левая
            if (((x+dx)>x1) &&(x<=x1) && (y<=y2) &&(y>=y1)) { 
                M(j + N_j * (i),j + N_j * (i)) = 1;//i, j
                R(j + N_j * (i)) = 100;
                continue;
            }

            if ((x>=x1) && (x<=x2) && (y<=y2) && (y>=y1)) {
				M(j + N_j * (i),j + N_j * (i)) = 1;//i, j
                R(j + N_j * (i)) = 100;
                continue;
            }
            if ((x<=x1) || (x>=x2) || ((x>x1 && x<x2) && ((y<=y1) || (y>=y2)))) { //%Основная часть
                M(j+N_j*(i),j+N_j*(i+1)) =a1/(dx*dx); //i+1, j
                M(j+N_j*(i),j+N_j*(i)) = -2*a1/(dx*dx)-2*a2/(dy*dy);//i,j
                M(j+N_j*(i),j+N_j*(i-1)) = a1/(dx*dx);//i-1, j,
                M(j+N_j*(i),j+N_j*(i)+1) =a2/(dy*dy);//i,j+1
                M(j+N_j*(i),j+N_j*(i)-1) =a2/(dy*dy);//i,j-1
                continue;
            }
            M(j+N_j*(i),j+N_j*(i)) = 1;//i,j
            R(j+N_j*(i)) = 0;
        }
    }

    matrix<double> iM (N_i*N_j,N_i*N_j);
    bool flag = false;
    iM = gjinverse (M, flag);
    vector<double> T1 = prod(iM,R);

    matrix<double> z1(N_i,N_j);
    matrix<double> x_f(N_i,N_j);
    matrix<double> y_f(N_i,N_j);

    for (int i = 0; i < N_i; i++) {
        for (int j = 0; j < N_j; j++) {
            x_f(i,j)=i*dx;
            y_f(i,j)=j*dy;
            z1(i,j)=T1(i*N_j+j);
        }
    }

    return z1;
}


int main() {
    double a1 = 2;
    double a2 = 2;
    double L1 = 12;
    double L2 = 6;
   	int N_i = 14;
	int N_j = 14;

    /*clock_t start = clock();*/

    matrix<double> z1 = all_calc(a1,a2,L1,L2, N_i, N_j, 1);
    matrix<double> z2 = all_calc(a1,a2,L1,L2, N_i*2, N_j*2, 1);

   /* clock_t end = clock();
    double seconds = (double)(end - start) / CLOCKS_PER_SEC;
    printf("The time: %f seconds\n", seconds); */
    return 1;
} 
